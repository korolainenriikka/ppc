#include <iostream>
#include <vector>
#include <cstdlib>
#include <hip/hip_runtime.h>

struct Result {
    int y0;
    int x0;
    int y1;
    int x1;
    float outer[3];
    float inner[3];
};

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

/**
 * x, y: upper left corner of rectangle
 * c: color component
 * size_x, size_y: size of rectangle
*/
int inner_sum(int x, int y, int size_x, int size_y, int nx, int* sums) {
    // first orange square: x at x + size_x - 1,  y at y + size_y - 1 (-1 bc size starts from 1)
    int sum1_i = (x+size_x-1) + nx * (y+size_y-1);
    int sum1 = sums[sum1_i];

    // first blue square: x at x + size_x, y at y-1
    int sum2 = y > 0 ? sums[(x+size_x-1) + nx * (y-1)] : 0;
    
    // second blue square: x at x - 1, y at y + size_y
    int sum3 = x > 0 ? sums[(x-1) + nx * (y + size_y -1)] : 0;

    // last orange square
    int sum4 = (x > 0 && y > 0) ? sums[(x-1) + nx * (y-1)] : 0;

    int inner_sum = sum1 - sum2 - sum3 + sum4;
    return inner_sum;
}

__global__ void kernel(int* best_coords, float* best_sses, int* sums, int nx, int ny) {
    int size_x = threadIdx.x + blockIdx.x * blockDim.x;
    int size_y = threadIdx.y + blockIdx.y * blockDim.y;

    if (size_x > nx || size_y > ny || (size_x == nx && size_y == ny) || size_x == 0 || size_y == 0)
        return;

    int rec_size = size_y * size_x;
    int outer_size = nx*ny - rec_size;
    int total_sum = sums[nx*ny-1];

    float best_sse = total_sum;
    int best_x = 0;
    int best_y = 0;
    for (int y = 0; y <= ny - size_y; ++y) {
        for (int x = 0; x <= nx - size_x; ++x) {
            // inclusion/exclusion
            int sum1_i = (x+size_x-1) + nx * (y+size_y-1);
            int sum1 = sums[sum1_i];
            int sum2 = y > 0 ? sums[(x+size_x-1) + nx * (y-1)] : 0;
            int sum3 = x > 0 ? sums[(x-1) + nx * (y + size_y -1)] : 0;
            int sum4 = (x > 0 && y > 0) ? sums[(x-1) + nx * (y-1)] : 0;
            int in_sum = sum1 - sum2 - sum3 + sum4;

            int out_sum = total_sum - in_sum;

            // find inner and outer sse
            float sse_inner = in_sum * (1 - ((1.0/rec_size) * in_sum));
            float sse_outer = out_sum * (1 - ((1.0/outer_size) * out_sum));
            float sse = sse_inner + sse_outer;

            // compare sse to current minimum
            if (sse < best_sse) {
                best_sse = sse;
                best_y = y;
                best_x = x;
            }
        }
    }

    // write found solution to results
    // write y
    best_coords[0 + 2 * (size_x-1) + 2 * nx * (size_y-1)] = best_y;
    // write x
    best_coords[1 + 2 * (size_x-1) + 2 * nx * (size_y-1)] = best_x;
    // write sse
    best_sses[(size_x-1) + nx * (size_y-1)] = best_sse;
}


/*
This is the function you need to implement. Quick reference:
- x coordinates: 0 <= x < nx
- y coordinates: 0 <= y < ny
- color components: 0 <= c < 3
- input: data[c + 3 * x + 3 * nx * y]
*/
Result segment(int ny, int nx, const float *data) {
    // PREPROCESSING: create size nx*ny*3 array where each pixel is sum of that color component until that point
    std::vector<int> sums(nx*ny, 0);
    for (int y = 0; y < ny; ++y) {
        int row_sum = 0;
        for (int x = 0; x < nx; ++x) {
            int c = 0; // only read first color component
            int i = c + 3 * x + 3 * nx * y;
            int data_i = static_cast<int>(data[i]);
            row_sum += data_i;
            int sum_i_above = y == 0 ? 0 : sums[x + nx*(y-1)];
            sums[x+nx*y] = sum_i_above + row_sum;
        }
    }

    // Allocate GPU memory
    int* sumsGPU = NULL;
    CHECK(hipMalloc((void**)&sumsGPU, nx*ny * sizeof(int)));
    CHECK(hipMemcpy(sumsGPU, sums.data(), nx*ny * sizeof(int), hipMemcpyHostToDevice));
    
    std::vector<int> init_coords(2*nx*ny, 1111);
    int* coordGPU = NULL;
    CHECK(hipMalloc((void**)&coordGPU, 2*nx*ny * sizeof(int)));
    CHECK(hipMemcpy(coordGPU, init_coords.data(), 2*nx*ny * sizeof(int), hipMemcpyHostToDevice));

    float* sseGPU = NULL;
    CHECK(hipMalloc((void**)&sseGPU, nx*ny * sizeof(float)));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(nx, dimBlock.x), divup(ny, dimBlock.y));
    kernel<<<dimGrid, dimBlock>>>(coordGPU, sseGPU, sumsGPU, nx, ny);
    CHECK(hipGetLastError());

    // Copy data back to CPU
    std::vector<int> best_coords(2*nx*ny, 0);
    CHECK(hipMemcpy(best_coords.data(), coordGPU, 2*nx*ny * sizeof(int) , hipMemcpyDeviceToHost));
    
    std::vector<float> best_sses(nx*ny, 0.0);
    // -1: we do not write on the GPU to the last location as inner rec cannot be full rec
    CHECK(hipMemcpy(best_sses.data(), sseGPU, (nx*ny-1)*sizeof(float), hipMemcpyDeviceToHost));

    // Release memory
    CHECK(hipFree(sumsGPU));
    CHECK(hipFree(coordGPU));
    CHECK(hipFree(sseGPU));

    // POSTPROCESSING

    // find best segmentation
    float min_sse = 600*600;
    // int best_size_x = 0;
    // int best_size_y = 0;
    Result best_result{0, 0, 0, 0, {0, 0, 0}, {0, 0, 0}};
    // benchmark 3.txt: y=ny, x=1 kohdassa ollaan muistissa paikassa johon ei olla kirjotettu.
    // siellä sse on pieni, ja systeemi luulee että jess hyvä segmentointi ja palauttaa sen.
    // jos laittaa size_y < ny, menee koordinaattien suhteen läpi. sillon inner/outer sum laskenta jostain syystä timeouttaa?
    // mut muut testit failaa, jos muuttaa ton <= pois (esim small simple 1)
    for (int size_y = 1; size_y <= ny; ++size_y) {
        for (int size_x = 1; size_x <= nx; ++size_x) {
            if (size_x == nx && size_y == ny) {
                continue;
            }
            
            float sse_for_size = best_sses[(size_x-1) + nx*(size_y-1)];
            if (sse_for_size < min_sse) {
                min_sse = sse_for_size;
                int y0 = best_coords[0 + 2 * (size_x-1) + 2 * nx * (size_y-1)];
                int x0 = best_coords[1 + 2 * (size_x-1) + 2 * nx * (size_y-1)];
                best_result.y0 = y0; 
                best_result.x0 = x0;
                best_result.y1 = y0 + size_y;
                best_result.x1 = x0 + size_x;
                // best_size_x = size_x;
                // best_size_y = size_y;
            }
        }
    }
    
    // find averages for best segmentation
    // int min_size_x = best_result.x1 - best_result.x0;
    // int min_size_y = best_result.y1 - best_result.y0;
    // int rec_size = min_size_x * min_size_y;
    // int outer_size = nx*ny - rec_size;
    // int in_sum = inner_sum(best_result.x0, best_result.y0, min_size_x, min_size_y, nx, sums.data());
    // float inner_avg = in_sum / rec_size;

    // int out_sum = sums[nx*ny-1] - in_sum;
    // float outer_avg = out_sum / outer_size;

    // for (int i = 10; i < 1000; ++i) {
    //     std::cout << best_sses[i] << ' ';
    // }
    // paras alkupiste on (291, 85). tosi monella koolla alusta, kaikilla koilla, toi on oikein.)
    // koko jonka löydän on (400,1) (y,x). ihan väärin.

    float inner_avg = 0.0;
    float outer_avg = 0.0;

    best_result.outer[0] = outer_avg;
    best_result.outer[1] = outer_avg;
    best_result.outer[2] = outer_avg;
    best_result.inner[0] = inner_avg;
    best_result.inner[1] = inner_avg;
    best_result.inner[2] = inner_avg;

    return best_result;
}