
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>

void correlate(int ny, int nx, const float *data, float *result) {
    // NORMALIZATION
    std::vector<double> normalized(ny*nx, 0.0);

    // vähennä mean jokaisesta elementistä
    for (int y = 0; y < ny; ++y) {
        double sum = 0;
        for (int x = 0; x < nx; ++x) {
            sum += data[x + y*nx];
        }
        double mean = sum/nx;
    
        for (int x = 0; x < nx; ++x) {
            if (mean != 0) {
                normalized[x + y*nx] = data[x + y*nx] - mean;
            } else {
                normalized[x + y*nx] = data[x + y*nx];
            }
        }
    }

    // normalize the input rows so that for each row the sum of the squares of the elements is 1
    for (int y = 0; y < ny; ++y) {
        double sum_of_squares = 0.0;
        for (int x = 0; x < nx; ++x) {
            sum_of_squares += normalized[x + y*nx]*normalized[x + y*nx];
        }
        double sqrt_sum_of_squares = sqrt(sum_of_squares);

        for (int x = 0; x < nx; ++x) {
            normalized[x + y*nx] = normalized[x + y*nx] / sqrt_sum_of_squares;
        }
    }
    
    // MATRIX PRODUCT
    // diagonal elements are always 1
    for (int a = 0; a < ny; ++a) {
        result[a + a*ny] = 1;
    }

    // Calculate the (upper triangle of the) matrix product Y = XX^T
    for (int y = 0; y < ny; ++y) {
        for (int x = y+1; x < ny; ++x) {
            double dot_product = 0.0;
            for (int k = 0; k < nx; ++k) {
                dot_product += normalized[k + y*nx] * normalized[k + x*nx];
            }
            result[x + y * ny] = dot_product;
        }
    }
}
