#include <cmath>
#include <vector>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

__global__ void kernel(float* result, float* normalized_T, int ny, int nx) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= ny || y >= ny) {
        return;
    }

    if (x < y) {
        result[x+y*ny] = 0;
        return;
    }

    double dot_product = 0.0;
    for (int k = 0; k < nx; ++k) {
        dot_product += normalized_T[k*ny + y] * normalized_T[k*ny + x];
    }
    result[x + y * ny] = dot_product;
}

void correlate(int ny, int nx, const float *data, float *result) {
    // NORMALIZATION
    std::vector<float> normalized(ny*nx, 0.0);

    // vähennä mean jokaisesta elementistä
    for (int y = 0; y < ny; ++y) {
        float sum = 0;
        for (int x = 0; x < nx; ++x) {
            sum += data[x + y*nx];
        }
        float mean = sum/nx;
    
        for (int x = 0; x < nx; ++x) {
            if (mean != 0) {
                normalized[x + y*nx] = data[x + y*nx] - mean;
            } else {
                normalized[x + y*nx] = data[x + y*nx];
            }
        }
    }

    // normalize the input rows so that for each row the sum of the squares of the elements is 1
    for (int y = 0; y < ny; ++y) {
        float sum_of_squares = 0.0;
        for (int x = 0; x < nx; ++x) {
            sum_of_squares += normalized[x + y*nx]*normalized[x + y*nx];
        }
        float sqrt_sum_of_squares = sqrt(sum_of_squares);

        for (int x = 0; x < nx; ++x) {
            normalized[x + y*nx] = normalized[x + y*nx] / sqrt_sum_of_squares;
        }
    }
    std::vector<float> normalized_T(ny*nx, 0.0);
    for (int y = 0; y < ny; ++y) {
        for (int x = 0; x < nx; ++x) {
            normalized_T[y + x*ny] = normalized[x + y*nx];
        }
    }

    // Copy data GPU
    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, nx * ny * sizeof(float)));
    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, ny * ny * sizeof(float))); // result size is ny x ny
    // CHECK(cudaMemset((void**)&rGPU, 0, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(dGPU, normalized_T.data(), nx * ny * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    kernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, ny, nx);
    CHECK(hipGetLastError());

    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, rGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(rGPU));
}